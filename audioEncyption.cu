#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <sndfile.h>

typedef float DTYPE;

#define SHARED_MEM_SIZE 256
#define SAMPLING_FREQUENCY 44100
#define CARRIER_FREQUENCY 12800

bool loadAudioFile(const char *file_path, DTYPE **audio_data, unsigned int *audio_size)
{
    SNDFILE *snd_file;
    SF_INFO sf_info;

    snd_file = sf_open(file_path, SFM_READ, &sf_info);
    if (!snd_file)
    {
        fprintf(stderr, "Error opening file: %s\n", file_path);
        return false;
    }

    *audio_size = sf_info.frames * sf_info.channels;
    *audio_data = (DTYPE *)malloc(sizeof(DTYPE) * (*audio_size));

    sf_readf_float(snd_file, *audio_data, *audio_size);
    sf_close(snd_file);

    return true;
}

void saveAudioFile(const char *file_path, DTYPE *audio_data, unsigned int audio_size, unsigned int sample_rate)
{
    SNDFILE *snd_file;
    SF_INFO sf_info;

    sf_info.samplerate = sample_rate;
    sf_info.channels = 1; // Assuming mono audio
    sf_info.format = SF_FORMAT_WAV | SF_FORMAT_PCM_16;

    snd_file = sf_open(file_path, SFM_WRITE, &sf_info);
    sf_writef_float(snd_file, audio_data, audio_size);
    sf_close(snd_file);
}

__global__ void processAudio(DTYPE *input_audio, DTYPE *output_audio, unsigned int audio_size, bool encrypt)
{
    __shared__ DTYPE shared_buffer[SHARED_MEM_SIZE + 1];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    DTYPE accumulator = 0.0;

    for (int i = tid; i < audio_size; i += blockDim.x)
    {
        shared_buffer[threadIdx.x] = input_audio[i];
        __syncthreads();

        if (encrypt)
        {
            accumulator += sin(2.0 * M_PI * CARRIER_FREQUENCY * i / SAMPLING_FREQUENCY) * shared_buffer[threadIdx.x];
        }
        else
        {
            accumulator += sin(2.0 * M_PI * CARRIER_FREQUENCY * i / SAMPLING_FREQUENCY) * shared_buffer[threadIdx.x];

        }

        __syncthreads();
    }

    output_audio[tid] = accumulator;
}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        fprintf(stderr, "Usage: %s <cypher|decypher> <input.wav> <output.wav>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char *mode = argv[1];
    const char *input_file = argv[2];
    const char *output_file = argv[3];

    DTYPE *h_input_audio, *h_output_audio;
    unsigned int audio_size;

    if (!loadAudioFile(input_file, &h_input_audio, &audio_size))
    {
        fprintf(stderr, "Error loading input audio file\n");
        return EXIT_FAILURE;
    }

    h_output_audio = (DTYPE *)malloc(sizeof(DTYPE) * audio_size);

    DTYPE *d_input_audio, *d_output_audio;
    hipMalloc((void **)&d_input_audio, sizeof(DTYPE) * audio_size);
    hipMalloc((void **)&d_output_audio, sizeof(DTYPE) * audio_size);

    hipMemcpy(d_input_audio, h_input_audio, sizeof(DTYPE) * audio_size, hipMemcpyHostToDevice);

    unsigned int threads_per_block = 256;
    unsigned int num_blocks = (audio_size + threads_per_block - 1) / threads_per_block;

    // Call the GPU kernel for audio processing (encryption/decryption)
    processAudio<<<num_blocks, threads_per_block>>>(d_input_audio, d_output_audio, audio_size, strcmp(mode, "cypher") == 0);

    hipDeviceSynchronize();

    hipMemcpy(h_output_audio, d_output_audio, sizeof(DTYPE) * audio_size, hipMemcpyDeviceToHost);

    saveAudioFile(output_file, h_output_audio, audio_size, SAMPLING_FREQUENCY);

    free(h_input_audio);
    free(h_output_audio);

    hipFree(d_input_audio);
    hipFree(d_output_audio);

    return EXIT_SUCCESS;
}
